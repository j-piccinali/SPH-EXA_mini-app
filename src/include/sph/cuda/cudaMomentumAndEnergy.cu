#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "cudaMomentumAndEnergy.cuh"
#include "../kernels.hpp"

namespace sphexa
{
namespace sph
{

template void cudaComputeMomentumAndEnergy<double>(const std::vector<int> &l, SqPatch<double> &d);

const double gradh_i = 1.0;
const double gradh_j = 1.0;
const double ep1 = 0.2, ep2 = 0.02;
const int mre = 4;

#define CHECK_CUDA_ERR(errcode) checkErr((errcode), __FILE__, __LINE__, #errcode);

void checkErr(hipError_t err, const char *filename, int lineno, const char *funcName)
{
    if (err != hipSuccess)
    {
        const char *errName = hipGetErrorName(err);
        const char *errStr = hipGetErrorString(err);
        fprintf(stderr, "CUDA Error at %s:%d. Function %s returned err %d: %s - %s\n", filename, lineno, funcName, err, errName, errStr);
    }
}

template <typename T>
__global__ void momenumAndEnergy_manyParticlesPerBlock(const int n, const int dx, const T sincIndex, const T K, const int ngmax,
                                                       const BBox<T> *bbox, const int *clist, const int *neighbors,
                                                       const int *neighborsCount, const T *x, const T *y, const T *z, const T *vx,
                                                       const T *vy, const T *vz, const T *h, const T *m, const T *ro, const T *p,
                                                       const T *c, T *grad_P_x, T *grad_P_y, T *grad_P_z, T *du)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];

    T momentum_x = 0.0, momentum_y = 0.0, momentum_z = 0.0, energy = 0.0;

    T A_i = 0.0;
    if (p[i] < 0.0) A_i = 1.0;

    // int converstion to avoid a bug that prevents vectorization with some compilers
    for (int pj = 0; pj < nn; pj++)
    {
        const int j = neighbors[tid * ngmax + pj];

        // calculate the scalar product rv = rij * vij

        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);

        T v_ijx = (vx[i] - vx[j]);
        T v_ijy = (vy[i] - vy[j]);
        T v_ijz = (vz[i] - vz[j]);

        T rv = r_ijx * v_ijx + r_ijy * v_ijy + r_ijz * v_ijz;

        T r_square = (r_ijx * r_ijx) + (r_ijy * r_ijy) + (r_ijz * r_ijz);

        T r_ij = sqrt(r_square);
        T rv_i = r_ij / h[i];
        T rv_j = r_ij / h[j];

        T viscosity_ij = artificial_viscosity(ro[i], ro[j], h[i], h[j], c[i], c[j], rv, r_square);

        T derivative_kernel_i = wharmonic_derivative(rv_i, h[i], sincIndex, K);
        T derivative_kernel_j = wharmonic_derivative(rv_j, h[j], sincIndex, K);

        // divide by r_ij? missing h?
        T grad_v_kernel_x_i = r_ijx * derivative_kernel_i;
        T grad_v_kernel_y_i = r_ijy * derivative_kernel_i;
        T grad_v_kernel_z_i = r_ijz * derivative_kernel_i;

        T grad_v_kernel_x_j = r_ijx * derivative_kernel_j;
        T grad_v_kernel_y_j = r_ijy * derivative_kernel_j;
        T grad_v_kernel_z_j = r_ijz * derivative_kernel_j;

        T grad_v_kernel_x_ij = (grad_v_kernel_x_i + grad_v_kernel_x_j) / 2.0;
        T grad_v_kernel_y_ij = (grad_v_kernel_y_i + grad_v_kernel_y_j) / 2.0;
        T grad_v_kernel_z_ij = (grad_v_kernel_z_i + grad_v_kernel_z_j) / 2.0;

        T force_i_j_r = expf(-(rv_i * rv_i)) * expf((dx * dx) / (h[i] * h[i]));

        T A_j = 0.0;
        if (p[j] < 0.0) A_j = 1.0;

        T delta_pos_i_j = 0.0;
        if (p[i] > 0.0 && p[j] > 0.0) delta_pos_i_j = 1.0;

        T R_i_j = ep1 * (A_i * abs(p[i]) + A_j * abs(p[j])) + ep2 * delta_pos_i_j * (abs(p[i]) + abs(p[j]));

        T r_force_i_j = R_i_j * pow(force_i_j_r, (int)mre);

        T partial_repulsive_force = (r_force_i_j / (ro[i] * ro[j]));

        T pro_i = p[i] / (gradh_i * ro[i] * ro[i]);
        T pro_j = p[j] / (gradh_j * ro[j] * ro[j]);

        momentum_x +=
            m[j] * (pro_i * grad_v_kernel_x_i + pro_j * grad_v_kernel_x_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_x_ij);
        momentum_y +=
            m[j] * (pro_i * grad_v_kernel_y_i + pro_j * grad_v_kernel_y_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_y_ij);
        momentum_z +=
            m[j] * (pro_i * grad_v_kernel_z_i + pro_j * grad_v_kernel_z_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_z_ij);

        energy += m[j] * (pro_i + 0.5 * viscosity_ij) * (v_ijx * grad_v_kernel_x_i + v_ijy * grad_v_kernel_y_i + v_ijz * grad_v_kernel_z_i);
    }

    du[i] = energy;

    grad_P_x[i] = momentum_x;
    grad_P_y[i] = momentum_y;
    grad_P_z[i] = momentum_z;
}

template <typename T>
void cudaComputeMomentumAndEnergy(const std::vector<int> &l, SqPatch<T> &d)
{
    const size_t n = l.size();
    const size_t ngmax = d.ngmax;
    const size_t allNeighbors = n * ngmax;
    const T dx = d.dx;
    const T sincIndex = d.sincIndex;
    const T K = d.K;
    const BBox<T> bbox = d.bbox;
    const size_t np = d.x.size();

    const int *clist = l.data();
    int *d_clist, *d_neighbors, *d_neighborsCount; // d_ prefix stands for device
    T *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_h, *d_m, *d_ro, *d_p, *d_c;
    T *d_grad_P_x, *d_grad_P_y, *d_grad_P_z, *d_du;
    BBox<T> *d_bbox;

    const size_t size_np_T = np * sizeof(T);
    const size_t size_allNeighbors_int = allNeighbors * sizeof(int);
    const size_t size_n_T = n * sizeof(T);
    const size_t size_n_int = n * sizeof(int);
    const size_t size_bbox = sizeof(BBox<T>);

    // const float neighborsSizeInGB = size_allNeighbors_int * 1e-9;
    // const float memorySizeInGB = (2 * size_n_int + size_allNeighbors_int + size_bbox + 11 * size_np_T + 4 * size_n_T)*1e-9;
    // printf("CUDA: Total GPU memory usage: %.2fGB\n", memorySizeInGB);

    // input data
    CHECK_CUDA_ERR(hipMalloc((void **)&d_clist, size_n_int));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_neighbors, size_allNeighbors_int));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_neighborsCount, size_n_int));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_bbox, size_bbox));

    CHECK_CUDA_ERR(hipMalloc((void **)&d_x, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_y, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_z, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_vx, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_vy, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_vz, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_h, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_m, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_ro, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_p, size_np_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_c, size_np_T));

    // output data
    CHECK_CUDA_ERR(hipMalloc((void **)&d_grad_P_x, size_n_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_grad_P_y, size_n_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_grad_P_z, size_n_T));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_du, size_n_T));

    CHECK_CUDA_ERR(hipMemcpy(d_clist, clist, size_n_int, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_neighbors, d.neighbors.data(), size_allNeighbors_int, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_neighborsCount, d.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_bbox, &bbox, size_bbox, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vx, d.vx.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vy, d.vy.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vz, d.vz.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_p, d.p.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c, d.c.data(), size_np_T, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel [manyParticlesPerBlock] launch with %d blocks of %d threads\n", blocksPerGrid,
    //        threadsPerBlock);
    momenumAndEnergy_manyParticlesPerBlock<T><<<blocksPerGrid, threadsPerBlock>>>(
        n, dx, sincIndex, K, ngmax, d_bbox, d_clist, d_neighbors, d_neighborsCount, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p,
        d_c, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du);

    CHECK_CUDA_ERR(hipGetLastError());

    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_x.data(), d_grad_P_x, size_n_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_y.data(), d_grad_P_y, size_n_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_z.data(), d_grad_P_z, size_n_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.du.data(), d_du, size_n_T, hipMemcpyDeviceToHost));

    CHECK_CUDA_ERR(hipFree(d_clist));
    CHECK_CUDA_ERR(hipFree(d_neighborsCount));
    CHECK_CUDA_ERR(hipFree(d_neighbors));

    CHECK_CUDA_ERR(hipFree(d_x));
    CHECK_CUDA_ERR(hipFree(d_y));
    CHECK_CUDA_ERR(hipFree(d_z));
    CHECK_CUDA_ERR(hipFree(d_vx));
    CHECK_CUDA_ERR(hipFree(d_vy));
    CHECK_CUDA_ERR(hipFree(d_vz));
    CHECK_CUDA_ERR(hipFree(d_h));
    CHECK_CUDA_ERR(hipFree(d_m));
    CHECK_CUDA_ERR(hipFree(d_ro));
    CHECK_CUDA_ERR(hipFree(d_p));
    CHECK_CUDA_ERR(hipFree(d_c));

    CHECK_CUDA_ERR(hipFree(d_grad_P_x));
    CHECK_CUDA_ERR(hipFree(d_grad_P_y));
    CHECK_CUDA_ERR(hipFree(d_grad_P_z));
    CHECK_CUDA_ERR(hipFree(d_du));
}
} // namespace sph
} // namespace sphexa
